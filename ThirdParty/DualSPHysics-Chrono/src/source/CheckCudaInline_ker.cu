//HEAD_DSPH
/*
 <DUALSPHYSICS>  Copyright (c) 2020 by Dr Jose M. Dominguez et al. (see http://dual.sphysics.org/index.php/developers/). 

 EPHYSLAB Environmental Physics Laboratory, Universidade de Vigo, Ourense, Spain.
 School of Mechanical, Aerospace and Civil Engineering, University of Manchester, Manchester, U.K.

 This file is part of DualSPHysics. 

 DualSPHysics is free software: you can redistribute it and/or modify it under the terms of the GNU General Public License as published by
 the Free Software Foundation, either version 3 of the License, or (at your option) any later version. 

 DualSPHysics is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even the implied warranty of
 MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License for more details. 

 You should have received a copy of the GNU General Public License, along with DualSPHysics. If not, see <http://www.gnu.org/licenses/>. 
*/

/// \file CheckCudaInline_ker.cu \brief It is only used for checking compilation of CUDA code in headers.

#include "TypesDef.h"
#include "DualSphDef.h"
#include <hip/hip_runtime_api.h>

namespace cutesting{
#include "FunctionsBasic_iker.h"
#include "FunctionsGeo3d_iker.h"
#include "FunSphKernel_iker.h"
#include "FunctionsMath_iker.h"
#include "JCellSearch_iker.h"


}


